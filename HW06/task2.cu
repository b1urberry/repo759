// task2.cu
#include <iostream>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <random>
#include <chrono>


#include "scan.cuh"

int main(int argc, char** argv) {
    if (argc < 3) {
        std::cerr << "Usage: " << argv[0] << " <n> <threads_per_block>" << std::endl;
        return 1;
    }

    int n = atoi(argv[1]);
    int threads_per_block = atoi(argv[2]);

    // Allocate managed memory
    float *input, *output;
    hipMallocManaged(&input, n*sizeof(float));
    hipMallocManaged(&output, n*sizeof(float));

    std::mt19937 rng(static_cast<unsigned int>(std::time(nullptr))); // seed the random number generator
    std::uniform_real_distribution<float> dist(-1.0, 1.0); 
    // Fill input with random float numbers in range [-1, 1]
    for (int i = 0; i < n; ++i) {
        input[i] = dist(rng);
    }

    // Time the scan operation using CUDA events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    scan(input, output, n, threads_per_block);
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);

    // Print the last element of the array and the time taken
    std::cout << output[n-1] << std::endl;
    std::cout << elapsedTime << std::endl;

    hipFree(input);
    hipFree(output);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
