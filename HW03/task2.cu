
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hiprand/hiprand_kernel.h>

const int ARRAY_SIZE = 16;
const int BLOCK_SIZE = 8;

__global__ void computeArray(int *dA, int a) {
    int tid = threadIdx.x;  // Get the current thread's ID within its block.
    int bid = blockIdx.x;  // Get the current block's ID.
    
    int idx = bid * BLOCK_SIZE + tid;
    
    int x = tid;
    int y = bid;

    dA[idx] = a * x + y;
}

int main() {
    int hA[ARRAY_SIZE]; 
    int *dA;
    int a;

    // Generate a random integer for 'a'
    a = rand() % 10; // Generate a random number between 0 and 9 for simplicity.

    printf("Randomly generated a: %d\n", a);

    // Allocate memory on the GPU.
    hipMalloc(&dA, ARRAY_SIZE * sizeof(int));

    // Launch the kernel with 2 blocks and 8 threads per block.
    computeArray<<<2, BLOCK_SIZE>>>(dA, a);

    // Copy results from device to host.
    hipMemcpy(hA, dA, ARRAY_SIZE * sizeof(int), hipMemcpyDeviceToHost);

    // Print the results.
    for (int i = 0; i < ARRAY_SIZE; i++) {
        printf("%d ", hA[i]);
    }
    printf("\n");

    // Free GPU memory.
    hipFree(dA);

    return 0;
}
