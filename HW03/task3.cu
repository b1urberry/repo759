// task3.cu

#include <stdio.h>
#include <hiprand/hiprand_kernel.h>
#include "vscale.cuh"

const int TPB = 16;  // Threads Per Block

int main(int argc, char *argv[]) {
    if (argc != 2) {
        return 1;
    }

    int n = atoi(argv[1]);

    float *a, *b, *d_a, *d_b;
    a = (float *)malloc(n * sizeof(float));
    b = (float *)malloc(n * sizeof(float));

    // Fill a and b with random numbers
    for (int i = 0; i < n; i++) {
        a[i] = ((float)rand() / RAND_MAX) * 20.0 - 10.0;  // Values between -10 and 10
        b[i] = (float)rand() / RAND_MAX;  // Values between 0 and 1
    }

    hipMalloc(&d_a, n * sizeof(float));
    hipMalloc(&d_b, n * sizeof(float));

    hipMemcpy(d_a, a, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, n * sizeof(float), hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int numBlocks = (n + TPB - 1) / TPB;

    hipEventRecord(start);
    vscale<<<numBlocks, TPB>>>(d_a, d_b, n);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    hipMemcpy(b, d_b, n * sizeof(float), hipMemcpyDeviceToHost);

    float ms;
    hipEventElapsedTime(&ms, start, stop);

    printf("%f\n%f\n%f\n", ms, b[0], b[n-1]);

    free(a);
    free(b);
    hipFree(d_a);
    hipFree(d_b);

    return 0;
}
