#include <iostream>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <random>
#include <chrono>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>


int main(int argc, char** argv) {

    if (argc < 2) {
        std::cerr << "Usage: " << argv[0] << "n" << std::endl;
        return 1;
    }

    int n = atoi(argv[1]);

    // create the host vector
    thrust::host_vector<float> H(n);
    
    // fill the host vector H with random float numbers in range [-1, 1]
    std::mt19937 rng(static_cast<unsigned int>(std::time(nullptr))); // seed the random number generator
    std::uniform_real_distribution<float> dist(-1.0, 1.0); 
    for (int i = 0; i < n; ++i) {
        H[i] = dist(rng);
    }

    thrust::device_vector<float> D = H;

    // Time the scan operation using CUDA events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    float result = thrust::reduce(D.begin(), D.end());
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);

    // Print the last element of the array and the time taken
    std::cout << result << std::endl;
    std::cout << elapsedTime << std::endl;

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
