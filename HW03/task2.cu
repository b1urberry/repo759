
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hiprand/hiprand_kernel.h>

const int ARRAY_SIZE = 16;
const int BLOCK_SIZE = 8;

__global__ void computeArray(int *dA, int a) {
    int tid = threadIdx.x;  // Get the current thread's ID within its block.
    int bid = blockIdx.x;  // Get the current block's ID.
    
    int idx = bid * BLOCK_SIZE + tid;
    
    int x = tid;
    int y = bid;

    dA[idx] = a * x + y;
}

int main() {
    int hA[ARRAY_SIZE]; // allocate an array of 16 ints on the device called dA from the host
    int *dA;

    // Generate a random integer for 'a'
    int a = rand() % 10; // Generate a random number between 0 and 9 for simplicity.

    hipMalloc(&dA, ARRAY_SIZE * sizeof(int));

    // Launch the kernel with 2 blocks and 8 threads per block.
    computeArray<<<2, BLOCK_SIZE>>>(dA, a);

    // Copies back the data stored in the device array dA into a host array called hA.
    hipMemcpy(hA, dA, ARRAY_SIZE * sizeof(int), hipMemcpyDeviceToHost);

    // Print (from the host) the 16 values stored in the host array separated by a single space each.
    for (int i = 0; i < ARRAY_SIZE; i++) {
        printf("%d ", hA[i]);
    }
    printf("\n");

    // Free GPU memory.
    hipFree(dA);

    return 0;
}
